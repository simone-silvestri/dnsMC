#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <time.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include "param.h"
#include "definitions.h"

#include <iostream>
#include <vector>
#include <algorithm>
#include "functions.h"
#include "hip/device_functions.h"
#include "memory.h"

using namespace std;
const int Ntot = imax*jmax*kmax;
const int N = Ntot/num_streams/num_gpu/p_row;
myfloat kP[nT] ,prob[nT][nB] ,probg[nT][nB][nQ];
myfloat kPp[nT],probp[nT][nB];

// global declaration of 2D float texture (visible for host and device code)
texture<myfloat, hipTextureType2D, hipReadModeElementType> tex_Ib;
__device__ __constant__ myfloat probd[2][nB];
__device__ myfloat solution[num_streams][N];
__device__ myfloat variance[num_streams][N];
#if srt==1
void sort_idx(NarrowBand *narrBand);
#endif

// pointer definitions
Gridn *gridGPU[num_gpu];
EmissSpec *Ibw_d[num_gpu];
hipArray *cuArray[num_gpu];
hipTextureObject_t *tex_tempf_d[num_gpu];
int *idx_d[num_gpu];
myfloat *wvc_d[num_gpu];
myfloat *Tnb_d[num_gpu];
hipTextureObject_t *tex_d[num_gpu];
hipTextureObject_t *tex_prob_d[num_gpu];


__global__ void kernel_fluid(Gridn *my_grid, myfloat *wvc_d, myfloat *Tnb_t, int n, int ns, int stream, myfloat kappamax, myfloat Tmax, int *idx_nb,
		hipTextureObject_t *tex_Tf, hipTextureObject_t *tex, hipTextureObject_t *tex_prob,
		int gpu, int ystart, EmissSpec *Ibw)
{

	Count cnt;

	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	volatile int tx = threadIdx.x;

	// for the moment ray stays in the register memory, if it spills,
	// we have to move it to the shared memory (it should fit in register memory)
	Beam ray;
	// we start defining the ray-Ibmax that is equal for every position

	__const__ myfloat Tnb[] = {Tnb_t[1] - Tnb_t[0], (Tnb_t[0])/(Tnb_t[1] - Tnb_t[0])};
	myfloat abscotemp1,abscotemp2;
	myfloat ratioI,invTi4;
	myfloat pos;
	myfloat Ibmax = 4.0f * kappamax * pow4(Tmax) * stefan / photmax;


	//define a narrow band
	myfloat wvc;
	__shared__ hiprandState_t state[blockdim];
	// this is basically the loop over the indices in the CPU.
	// the indices are found in relation with the stream-block-Thread design.
	// this is a grid-stride loop, hence every thread will do multiple cells
	for (int idx = tid; idx < n; idx += blockDim.x * gridDim.x)
	{
		solution[stream][idx] = 0;
		ray.i = (idx / (kmax*jmax/num_gpu/p_row) + 1) + (stream)*imax/ns;
		ray.j = (idx / (kmax/num_gpu) + 1 - (ray.i-1-(stream)*imax/ns)*jmax/p_row);
		ray.k = (idx - (kmax/num_gpu) * (ray.j-1 + (ray.i-1-(stream)*imax/ns)*jmax/p_row) + 1) + gpu*kmax/num_gpu;
		ray.j+= ystart-1;

		ray.Ti = tex3D<myfloat>(tex_Tf[0], ray.i+0.5f, ray.j+0.5f, ray.k+0.5f);

		myfloat De_var[nVar];

		for (int v = 0; v<nVar; v++)
		{
			De_var[v] = 0;
			int g=0;
			int nb=0;

			hiprand_init(1234, (idx+1)*(stream+1)*(v+1)*(1+gpu)*ystart, 0, &state[tx]);  // 	Initialize CURAND
#if srt == 1
			kernel_find(&cnt, Tnb_t, Tmax, &state[tx], tex_prob);
#endif
			int countnb=0;
			int countg=0;

			myfloat De_OERMc=0;
			/***********  ENTERING THE PHOTON LOOP, DEVIDED INTO SUB-BUNDLES *****************/

			for (int h=0; h<photmax; h++)
			{
				ray.xp = my_grid[0].x[ray.i];
				ray.yp = my_grid[0].y[ray.j];
				ray.zp = my_grid[0].z[ray.k];
				ray.ic = ray.i;
				ray.jc = ray.j;
				ray.kc = ray.k;
				int flag[3];
				// Now we have to deal with random numbers, selecting the angles
				// define the scattering function
				emiss_ang(&ray, &state[tx]);

				if(ray.sx == 0)
					ray.sx = 1e-10;
				if(ray.sy == 0)
					ray.sy = 1e-10;
				if(ray.sz == 0)
					ray.sz = 1e-10;
				flag[0] = (int)(ray.sx<0);
				flag[1] = (int)(ray.sy<0);
				flag[2] = (int)(ray.sz<0);

				//Find the narrowband index and the g index, based on random or based on the previous count
#if srt == 1
				wave_find(&nb,&g,&countnb,&countg,idx_nb,&cnt);
#else
				// non sorted monte carlo
				int tm =  (int) ((Tmax - Tnb_t[0])/(Tnb_t[1]-Tnb_t[0]));
				find_band(Tnb_t,Tmax,&g,&nb,&state[tx],tm, tex_prob);
#endif
				// the ray emanating from cell i,j,k is completely defined now
				// now we have to march it
				// initializing transmissivity

				ray.tra = 1.0f;
				wvc  =  wvc_d[nb];

				Emission emiss;

				// wall emissions
				emiss.west  =  Ibw->west[nb];
				emiss.east  =  Ibw->east[nb];
				//    			emiss.north =  Ibw->north[nb];
				//    			emiss.south =  Ibw->south[nb];
				//    			emiss.top   =  Ibw->top[nb];
				//    			emiss.bot   =  Ibw->bot[nb];

				// Calculate parameters for calculation of De_OERMc in march_ray
				pos = ((ray.Ti - Tnb_t[0])/(Tnb_t[1]-Tnb_t[0]));
				abscotemp2 = tex3D<myfloat>(*tex, nb+0.5f, pos+0.5f, g+0.5f);
				invTi4 = 1.0f / tex2D(tex_Ib,nb+0.5f ,pos+0.5f);

				pos = ((Tmax - Tnb_t[0])/(Tnb_t[1]-Tnb_t[0]));
				abscotemp1 = tex3D<myfloat>(*tex, nb+0.5f, pos+0.5f, g+0.5f);
				ratioI = (1.0f / invTi4) / tex2D(tex_Ib,nb+0.5f ,pos+0.5f) * abscotemp2 / abscotemp1 ;

				// Adjust the cells according to the scattering of the ray
				// if the ray originates from the negative cell boundary and it
				// is going in the negative direction, shift cells
				if ( flag[0] && ray.xp == my_grid[0].xu[ray.ic - 1] )
					ray.ic = ray.ic - 1;
				if ( flag[1] && ray.yp == my_grid[0].yu[ray.jc - 1] )
					ray.jc = ray.jc - 1;
				if ( flag[2] && ray.zp == my_grid[0].zu[ray.kc - 1] )
					ray.kc = ray.kc - 1;
				// Do the same thing if it is happening in the positive direction
				if ( !flag[0] && ray.xp == my_grid[0].xu[ray.ic] )
					ray.ic = ray.ic + 1;
				if ( !flag[1] && ray.yp == my_grid[0].yu[ray.jc] )
					ray.jc = ray.jc + 1;
				if ( !flag[2] && ray.zp == my_grid[0].zu[ray.kc] )
					ray.kc = ray.kc + 1;

				/**************************************************************************************/
				/************************** ENTERING THE MARCHING LOOP ********************************/
				/**************************************************************************************/

				//loop on all grids with maximum counter my_grid[grd].sm
				for(int grd = 0; grd < grid_num; grd++)
				{
					march_ray(&ray, nb, g, flag, &De_OERMc, Ibmax, my_grid[grd], ratioI, invTi4,
							tex_Tf[grd], emiss, wvc, Tnb, tex);

					if(grd<grid_num-1) {
						//adapting grid
						ray.ic += my_grid[grd].im/my_grid[grd+1].im-1;
						ray.jc += my_grid[grd].jm/my_grid[grd+1].jm-1;
						ray.kc += my_grid[grd].km/my_grid[grd+1].km-1;
						ray.ic /= my_grid[grd].im/my_grid[grd+1].im;
						ray.jc /= my_grid[grd].jm/my_grid[grd+1].jm;
						ray.kc /= my_grid[grd].km/my_grid[grd+1].km;
					}
				};

			}

			/***********  OUT OF THE PHOTON LOOP ****************************/
			solution[stream][idx] += De_OERMc;
			De_var[v] = De_OERMc;
		}
		/***********  OUT OF THE VARIANCE LOOP ****************************/

		solution[stream][idx] /= nVar;
		variance[stream][idx] = 0;
		for (int v = 0; v<nVar; v++)
		{
			variance[stream][idx] += 1./(nVar-0.5) * 1./(nVar-1) * (De_var[v] - solution[stream][idx]) * (De_var[v] - solution[stream][idx]);
		}
	}
	/***********  OUT OF THE GRID-STRIDE LOOP ****************************/

}

__device__ __forceinline__ void march_ray(Beam *ray, int nb, int g, int *flag, myfloat *De_OERMc,
		myfloat Ibmax, Gridn grid, myfloat ratioI, myfloat invTi4, hipTextureObject_t tex_Tf,
		Emission emiss, myfloat wvc, const myfloat Tnb[], hipTextureObject_t *tex)
{

	int counter = 0;
	while ( (ray->tra > toll) && (counter<grid.sm) )
	{

		// find the distance to cell boundary x,y,z -> the minimal will be the crossing
		myfloat dsx, dsy, dsz;
		dsx = ( grid.xu[ray->ic-flag[0]] - ray->xp ) / ray->sx;
		dsy = ( grid.yu[ray->jc-flag[1]] - ray->yp ) / ray->sy;
		dsz = ( grid.zu[ray->kc-flag[2]] - ray->zp ) / ray->sz;

		// interpolate to find the temperature of the cell (particle and fluid)
		myfloat tf = tex3D<myfloat>(tex_Tf,ray->ic+0.5f,ray->jc+0.5f,ray->kc+0.5f);

		// interpolate to find the absorption of the cell (particle and fluid)
		myfloat pos = (tf/Tnb[0] - Tnb[1]);
		myfloat absco =  tex3D<myfloat>(*tex, nb+0.5f, pos+0.5f, g+0.5f);

		// black-body radiation of the cell (particle and fluid)
		myfloat blackpow = tex2D(tex_Ib,nb+0.5f ,pos+0.5f);
		myfloat ds = dsx;
		(void) ( (ds > dsy) && (ds = dsy) );
		(void) ( (ds > dsz) && (ds = dsz) );

		// update ray position and scattering length
		(void) ( (ds==dsx) && (ray->xp = grid.xu[ray->ic-flag[0]]) );
		(void) ( (ds!=dsx) && (ray->xp =   ray->xp + ds * ray->sx) );
		(void) ( (ds==dsy) && (ray->yp = grid.yu[ray->jc-flag[1]]) );
		(void) ( (ds!=dsy) && (ray->yp =   ray->yp + ds * ray->sy) );
		(void) ( (ds==dsz) && (ray->zp = grid.zu[ray->kc-flag[2]]) );
		(void) ( (ds!=dsz) && (ray->zp =   ray->zp + ds * ray->sz) );

		myfloat alpha  = 1.0f - __expf(-ds * (absco));

		*De_OERMc -= Ibmax * ray->tra * alpha * ratioI *
				( invTi4 * blackpow - 1.0f );

		// update transmissivity of the ray and total distance travelled
		ray->tra = ray->tra * (1-alpha);
		if ( ray->tra < toll)
		{
			*De_OERMc -= Ibmax * ray->tra * ratioI *
					( invTi4 * blackpow - 1.0f );
			ray->tra = 0;
		}

		// Updating cell indices and Boundary conditions if end is reached
		// efficient ray tracing method?

		if ( dsx<dsy )
		{
			if ( dsx<dsz )
			{
				if ( flag[0] )
				{
					ray->ic = ray->ic-1;
					if ( ray->ic == 0 )
					{
						if(bdw==1)
						{
							ray->ic = grid.im;
							ray->xp = Lx;
						}
						else if(bdw==2)
						{
							*De_OERMc -= Ibmax * ray->tra*( invTi4 * emiss.west - 1.0f )* ratioI;
							ray->tra = 0;
						}
					}
				}
				else
				{
					ray->ic = ray->ic+1;
					if ( ray->ic == grid.im+1 )
					{
						if(bde==1)
						{
							ray->ic = 1;
							ray->xp = 0;
						}
						else if(bde==2)
						{
							*De_OERMc -= Ibmax * ray->tra*( invTi4 * emiss.east - 1.0f )* ratioI;
							ray->tra = 0;
						}
					}
				}
			}
			else
			{
				if ( flag[2] )
				{
					ray->kc = ray->kc-1;
					if ( ray->kc == 0 )
					{
						if(bdb==1)
						{
							ray->kc = grid.km;
							ray->zp = Lz;
						}
						else if(bdb==2)
						{
							*De_OERMc -= Ibmax * ray->tra*( invTi4 * emiss.bot - 1.0f )* ratioI;
							ray->tra = 0;
						}
					}
				}
				else
				{
					ray->kc = ray->kc+1;
					if ( ray->kc == grid.km+1 )
					{
						if(bdt==1)
						{
							ray->kc = 1;
							ray->zp = 0;
						}
						else if(bdt==2)
						{
							*De_OERMc -= Ibmax * ray->tra*( invTi4 * emiss.top - 1.0f )* ratioI;
							ray->tra = 0;
						}
					}
				}
			}
		}
		else
		{
			if ( dsy<dsz )
			{
				if ( flag[1] )
				{
					ray->jc = ray->jc-1;
					if ( ray->jc == 0 )
					{
						if(bds==1)
						{
							ray->jc = grid.jm;
							ray->yp = Ly;
						}
						else if(bds==2)
						{
							*De_OERMc -= Ibmax * ray->tra*( invTi4 * emiss.south - 1.0f )* ratioI;
							ray->tra = 0;
						}
					}
				}
				else
				{
					ray->jc = ray->jc+1;
					if ( ray->jc == grid.jm+1 )
					{
						if(bdn==1)
						{
							ray->jc = 1;
							ray->yp = 0;
						}
						else if(bdn==2)
						{
							*De_OERMc -= Ibmax * ray->tra*( invTi4 * emiss.north - 1.0f )* ratioI;
							ray->tra = 0;
						}
					}
				}
			}
			else
			{
				if ( flag[2] )
				{
					ray->kc = ray->kc-1;
					if ( ray->kc == 0 )
					{
						if(bdb==1)
						{
							ray->kc = grid.km;
							ray->zp = Lz;
						}
						else if(bdb==2)
						{
							*De_OERMc -= Ibmax * ray->tra*( invTi4 * emiss.bot - 1.0f )* ratioI;
							ray->tra = 0;
						}
					}
				}
				else
				{
					ray->kc = ray->kc+1;
					if ( ray->kc == grid.km+1 )
					{
						if(bdt==1)
						{
							ray->kc = 1;
							ray->zp = 0;
						}
						else if(bdt==2)
						{
							*De_OERMc -= Ibmax * ray->tra *( invTi4 * emiss.top - 1.0f )* ratioI;
							ray->tra = 0;
						}
					}
				}
			}
		}
		counter+=1;

	}  // closing the while statement
}

extern "C" void mc_gpu_(myfloatF *Tfort, int *ystart)
{

	myfloat kappamax,Tmax;

	/**************************************************************/
	/********** CREATING GRID AND TEMPERATURE FIELD ***************/
	/**************************************************************/

	Var_CPU varCPU[grid_num];
	Gridn gridCPU[grid_num];
	for (int grd = 0; grd < grid_num; grd++)
	{
		varCPU[grd].mk_grid(maxi[grd],maxj[grd],maxk[grd],maxi[0]);
		gridCPU[grd].mk_grid(maxi[grd],maxj[grd],maxk[grd],maxi[0],maxs[grd]);
	}
	for (int k = 0; k < (kmax+2); k++)
	{
		for (int j = 0; j < (jmax+2); j++)
		{
			for (int i = 0; i < (imax+2); i++)
			{
				varCPU[0].T[idx_T(i,j,k,imax,jmax)] = (myfloat) Tfort[idx_T(i,j,k,imax,jmax)];
			}
		}
	}

	// interpolating temperature and finding new concentration on coarser grid
	interp3D(varCPU);

	Tmax = 0;
	for (int k=1; k<kmax+1; k++)
	{
		for (int j=1; j<jmax+1; j++)
		{
			for (int i=1; i<imax+1; i++)
			{
				Tmax = MAX(Tmax,varCPU[0].T[idx_T(i,j,k,maxi[0],maxj[0])]);
			}
		}
	}
	Tmax = MAX(Tmax,Tww);
	Tmax = MAX(Tmax,Twe);
	Tmax = MAX(Tmax,Twn);
	Tmax = MAX(Tmax,Tws);
	Tmax = MAX(Tmax,Twt);
	Tmax = MAX(Tmax,Twb);

	/**************************************************************/
	/***********  READ THE TABLES *********************************/
	/**************************************************************/

	NarrowBand *narrBand;
	myfloat *Tnb;
	myfloat *prob_h, *probg_h, *prob_h2;

	prob_h   = (myfloat*)malloc(nB*nT*   sizeof(myfloat));
	probg_h  = (myfloat*)malloc(nB*nT*nQ*sizeof(myfloat));
	prob_h2  = (myfloat*)malloc(nB*2*sizeof(myfloat));

	narrBand = (NarrowBand*)malloc(nB *  sizeof(NarrowBand));
	Tnb = (myfloat*)malloc(nT*sizeof(myfloat));

	readT(narrBand, Tnb, kP, prob_h, probg_h, Tmax, &kappamax);

	/**************************************************************/
	/***********  FINISHED READING ********************************/
	/**************************************************************/

	//Sorting NarrowBands based on kavg of the band
#if srt == 1
	sort_idx(narrBand);
#endif

	/**************************************************************/
	/************* MEMORY COPY TO THE GPU's ***********************/
	/**************************************************************/

	hipStream_t streams0[num_streams];
	hipStream_t streams1[num_streams];

	for(int gpu = 0; gpu < num_gpu; gpu++) {
		hipSetDevice(gpu);

		for (int i=0; i<num_streams; i++)
		{
			if(gpu==0) {
				hipStreamCreate(&streams0[i]);
			}
			else {
				hipStreamCreate(&streams1[i]);
			}
		}

		// grid copy to GPU
		hipMalloc((void**)&gridGPU[gpu], grid_num * sizeof(Gridn));
		grid_copy(gridCPU, gridGPU[gpu]);
		// Wall emission copy to GPU
		hipMalloc((void**)&Ibw_d[gpu], nB * sizeof(EmissSpec));
		black_copy(Ibw_d[gpu],narrBand);

		myfloat Ib[nT][nB];

		for(int t = 0; t<nT; t++ )
			for(int nb = 0; nb<nB; nb++ )
			{
#if srt ==1
		Ib[t][nb] = I_blackC(Tnb[t],narrBand[narrBand[nb].idx].wvc);
#else
		Ib[t][nb] = I_blackC(Tnb[t],narrBand[nb].wvc);
#endif
			}

		// Create explicit channel description (could use an implicit as well)
		hipChannelFormatDesc DescIb = hipCreateChannelDesc<myfloat>();
		hipMallocArray(&cuArray[gpu], &DescIb, nB, nT);
		hipMemcpyToArray(cuArray[gpu], 0, 0, Ib, nB*nT*sizeof(myfloat), hipMemcpyHostToDevice);
		tex_Ib.addressMode[0] = hipAddressModeClamp;
		tex_Ib.addressMode[1] = hipAddressModeClamp;
		tex_Ib.filterMode = hipFilterModeLinear;
		tex_Ib.normalized = false;
		hipBindTextureToArray(tex_Ib, cuArray[gpu], DescIb);

		//textured memory copy of interpolated temperature
		hipMalloc((void**)&tex_tempf_d[gpu], grid_num*sizeof(hipTextureObject_t) );
		temp_fluid_copy(tex_tempf_d[gpu], varCPU);

		//memory copy of -> sorted index 			idx_d
		//				 -> central wavenumber 		wvc_d
		//				 -> discrete temperature	Tnb_d
		//				 -> textured asb coeff		tex_d
		//				 -> textured emiss prob		tex_prob_d
		//				 -> phase function prob		prob_A_d
		//				 -> textured part prob		tex_part_d
		//				 -> particle scatt coeff	Csca_d
		//				 -> particle abs coeff		Cabs_d
		hipMalloc((void**)&idx_d[gpu],nB * sizeof(int));
		hipMalloc((void**)&wvc_d[gpu],nB * sizeof(myfloat));
		hipMalloc((void**)&Tnb_d[gpu],nT*sizeof(myfloat));
		hipMalloc((void**)&tex_d[gpu], sizeof(hipTextureObject_t) );
		hipMalloc((void**)&tex_prob_d[gpu], sizeof(hipTextureObject_t) );
		narrowband_copy(narrBand, wvc_d[gpu], idx_d[gpu], tex_d[gpu], tex_prob_d[gpu], Tnb_d[gpu], Tnb);

		// CUDA memory allocation
		int tm = (int) ((Tmax - Tnb[0])/(Tnb[1]-Tnb[0]));
		for(int j = 0; j < 2; j++)
			for(int i = 0; i < nB; i++ )
			{
				prob_h2[idx_p(j,i)] = prob[tm+j][i];
			}
		hipMemcpyToSymbol(HIP_SYMBOL(probd), prob_h2 , nB*2*sizeof(myfloat) ,0,hipMemcpyHostToDevice);
		cudaCheckErrors("Malloc fail");

		/**************************************************************/
		/***********  STARTING CUDA ROUTINES **************************/
		/**************************************************************/

		/**************************************************************/
		/***************** FLUID MONTE CARLO **************************/
		/**************************************************************/



		for (int i=0; i<num_streams; i++)
		{
			if(gpu==0) {
				// launch one worker kernel per stream
				kernel_fluid<<<nblocks, blockdim, 0, streams0[i]>>>(gridGPU[gpu], wvc_d[gpu], Tnb_d[gpu], N, num_streams, i, kappamax, Tmax, idx_d[gpu],
						tex_tempf_d[gpu], tex_d[gpu], tex_prob_d[gpu], gpu, *ystart, Ibw_d[gpu]);
			}
			else {
				kernel_fluid<<<nblocks, blockdim, 0, streams1[i]>>>(gridGPU[gpu], wvc_d[gpu], Tnb_d[gpu], N, num_streams, i, kappamax, Tmax, idx_d[gpu],
						tex_tempf_d[gpu], tex_d[gpu], tex_prob_d[gpu], gpu, *ystart, Ibw_d[gpu]);
			}
		}
		cudaCheckErrors("Failed kernel execution");
	}

	//freeing all the CPU used variables (GPU is freed automatically by hipDeviceReset(); )
	for(int grd=0; grd<grid_num; grd++)
	{
		varCPU[grd].destroyVar();
		gridCPU[grd].destroyVar();
	}
	free(prob_h);
	free(probg_h);
	free(prob_h2);
	free(narrBand);
	free(Tnb);

}

extern "C" void get_results_(myfloatF resfort[(imax+2)*(jmax/p_row+2)*(kmax+2)], myfloatF varfort[(imax+2)*(jmax/p_row+2)*(kmax+2)])
{
	// clock_t start,end;
	// start = clock();
	myfloat *host[num_streams][num_gpu];
	myfloat *varh[num_streams][num_gpu];
	for (int i=0; i<num_streams; i++) {
		for(int gpu = 0; gpu < num_gpu; gpu++) {
			host[i][gpu] = (myfloat*)malloc(N*sizeof(myfloat));
			varh[i][gpu] = (myfloat*)malloc(N*sizeof(myfloat));
		}
	}
	myfloat *device0[num_streams];
	myfloat *device1[num_streams];
	myfloat *vard0[num_streams];
	myfloat *vard1[num_streams];
	hipStream_t streams0[num_streams];
	hipStream_t streams1[num_streams];

	for(int gpu = 0; gpu < num_gpu; gpu++) {
		hipSetDevice(gpu);
		hipDeviceSynchronize();
		for (int i=0; i<num_streams; i++)
		{
			if(gpu==0) {
				hipMalloc((void**)&device0[i]  ,N * sizeof(myfloat));
				cudaCheckErrors("Malloc fail device");
				hipMalloc((void**)&vard0[i]  ,N * sizeof(myfloat));
				cudaCheckErrors("Malloc fail device");
				hipStreamCreate(&streams0[i]);
			}
			else {
				hipMalloc((void**)&device1[i]  ,N * sizeof(myfloat));
				cudaCheckErrors("Malloc fail device");
				hipMalloc((void**)&vard1[i]  ,N * sizeof(myfloat));
				cudaCheckErrors("Malloc fail device");
				hipStreamCreate(&streams1[i]);
			}
		}


		for (int i=0; i<num_streams; i++)
		{
			if(gpu==0) {
				// launch one worker kernel per stream
				kernel_results<<<nblocks, blockdim, 0, streams0[i]>>>(vard0[i],device0[i], N, num_streams, i);
			}
			else {
				kernel_results<<<nblocks, blockdim, 0, streams1[i]>>>(vard1[i],device1[i], N, num_streams, i);
			}
		}
		cudaCheckErrors("Failed kernel execution");
	}
	for(int gpu = 0; gpu < num_gpu; gpu++) {
		hipSetDevice(gpu);
		for (int i = 0; i < num_streams; i++)
		{
			if(gpu==0) {
				hipMemcpyAsync(host[i][gpu],device0[i],N*sizeof(myfloat),hipMemcpyDeviceToHost,streams0[i]);
				cudaCheckErrors("Cuda memory copy asynchronous, device to host");
				hipMemcpyAsync(varh[i][gpu],vard0[i],N*sizeof(myfloat),hipMemcpyDeviceToHost,streams0[i]);
				cudaCheckErrors("Cuda memory copy asynchronous, device to host");
			}
			else {
				hipMemcpyAsync(host[i][gpu],device1[i],N*sizeof(myfloat),hipMemcpyDeviceToHost,streams1[i]);
				cudaCheckErrors("Cuda memory copy asynchronous, device to host");
				hipMemcpyAsync(varh[i][gpu],vard1[i],N*sizeof(myfloat),hipMemcpyDeviceToHost,streams0[i]);
				cudaCheckErrors("Cuda memory copy asynchronous, device to host");
			}
		}
		cudaCheckErrors("Copying to host fail");
	}

	cudaCheckErrors("unbind and/or free fail");

	/**************************************************************/
	/***************** RESETTING DEVICE MEMORY ********************/
	/**************************************************************/


	for(int gpu = 0; gpu < num_gpu; gpu++)
	{
		hipSetDevice(gpu);

#if (bdw==2) || (bde==2) || (bdn==2) || (bds==2) || (bdb==2) || (bdt==2)
		hipFree(Ibw_d[gpu]);
		cudaCheckErrors("unbind and/or free fail");
#endif
		hipFree(gridGPU[gpu]);
		cudaCheckErrors("unbind and/or free fail");
		hipFree(tex_tempf_d[gpu]);
		cudaCheckErrors("unbind and/or free fail");
		hipFree(idx_d[gpu]);
		cudaCheckErrors("unbind and/or free fail");
		hipFree(wvc_d[gpu]);
		cudaCheckErrors("unbind and/or free fail");
		hipFree(Tnb_d[gpu]);
		cudaCheckErrors("unbind and/or free fail");
		hipFree(tex_d[gpu]);
		cudaCheckErrors("unbind and/or free fail");
		hipFree(tex_prob_d[gpu]);
		cudaCheckErrors("unbind and/or free fail");
		hipFreeArray(cuArray[gpu]);
		cudaCheckErrors("unbind and/or free fail");

		for(int i=0; i<num_streams; i++) {
			if(gpu==0) {
				hipFree(device0[i]);
				hipFree(vard0[i]);
			}
			else {
				hipFree(device1[i]);
				hipFree(vard1[i]);
			}
		}
		hipDeviceReset();
	}

	/**************************************************************/
	/************ RETURNING RESULTS IN A 3D FASHION ***************/
	/**************************************************************/

	myfloat result[imax+2][jmax/p_row+2][kmax+2];

	int i, j, k;
	for(int stream = 0; stream < num_streams; stream++) {
		for(int n = 0; n < N; n++) {
			for(int gpu = 0; gpu < num_gpu; gpu++) {
				i = (n / (kmax*jmax/num_gpu/p_row) + 1) + (stream)*imax/num_streams;
				j = (n / (kmax/num_gpu) + 1 - (i-1-(stream)*imax/num_streams)*jmax/p_row);
				k = (n - (kmax/num_gpu) * (j-1 + (i-1-(stream)*imax/num_streams)*jmax/p_row) + 1) + gpu*kmax/num_gpu;
				result[i][j][k] = host[stream][gpu][n];
				resfort[idx_F(i,j,k)] = (myfloatF) result[i][j][k];
				varfort[idx_F(i,j,k)] = (myfloatF) powf(varh[stream][gpu][n],0.5);
			}
		}
	}
	for (int i=0; i<num_streams; i++) {
		for(int gpu = 0; gpu < num_gpu; gpu++) {
			free(host[i][gpu]);
			free(varh[i][gpu]);
		}
	}

}

__device__ __forceinline__ void kernel_find(Count *count, myfloat *Tnb, myfloat Tmax, hiprandState_t *state, hipTextureObject_t *tex_prob)
{
	int tm = (int) ((Tmax - Tnb[0])/(Tnb[1]-Tnb[0]));

	int nb = 0;
	int g = 0;
	for (int h = 0; h<nB; h++)
	{
		count->nb_cnt[h] = 0;
		for (int f = 0; f<nQ; f++)
		{
			count->g_cnt[f][h] = 0;
		}
	}
	/***********  ENTERING THE PHOTON LOOP ****************************/
	for (int h=0; h<photmax; h++)
	{
		// now we have to define the absorption narrow band wavenumber and the
		// quadrature point from prob and probg
		find_band(Tnb,Tmax,&g,&nb,state,tm,tex_prob);
		count->nb_cnt[nb] += 1;
		count->g_cnt[g][nb] +=1;

		/***********  OUT OF THE VARIANCE LOOP ****************************/
	}

}
__device__ __forceinline__ void emiss_ang(Beam *ray, hiprandState_t *state)
{
	myfloat phi   = hiprand_uniform(state)*2*pi;
	myfloat theta = acosf( 1 - 2*hiprand_uniform(state) );
	ray->sx = __cosf(theta);
	ray->sy = __sinf(theta)*__cosf(phi);
	ray->sz = __sinf(theta)*__sinf(phi);
}
__device__ __forceinline__ void find_band(myfloat *Tnb, myfloat Tmax, int *g, int *nb, hiprandState_t *state, int tm, hipTextureObject_t *tex_prob)
{

	myfloat Rwave = hiprand_uniform(state);
	// find index of temperature
	//temperature index is t and t+1, now search for R on t and t+1
	int t;
	if( (Tmax - Tnb[tm]) < (Tnb[tm+1] - Tmax) )
	{
		t = tm;
		int nb1 = 0;
		int w = nB;
		while (w > nb1+1)
		{
			int d = (nb1+w)/2;
			(void)((Rwave <  probd[t-tm][d] ) && (w=d));
			(void)((Rwave >= probd[t-tm][d] ) && (nb1=d));
		};
		nb1 = nb1+1;
		if( Rwave <  probd[t-tm][0] )
			nb1 = 0;
		if(Rwave == 1 )
			nb1 = nB -1;

		*nb = nb1;
	}
	else
	{
		t = tm+1;
		int nb2 = 0;
		int w = nB;
		while (w > nb2+1)
		{
			int d = (nb2+w)/2;
			(void)((Rwave <  probd[t-tm][d] ) && (w=d));
			(void)((Rwave >= probd[t-tm][d] ) && (nb2=d));
		};
		nb2 = nb2+1;
		if(Rwave <  probd[t-tm][0] )
			nb2 = 0;
		if(Rwave == 1 )
			nb2 = nB -1;
		*nb = nb2;
	}

#if grey == 1
	*g=0;
#else
	myfloat Rwave2 = hiprand_uniform(state);
	myfloat prob;

	*g =0;
	int w = nQ;
	while (w > *g+1)
	{
		int d = (*g+w)/2;
		prob = tex3D<myfloat>(*tex_prob, d+0.5f, *nb+0.5f, t+0.5f);
		(void)((Rwave2 < prob ) && ( w=d));
		(void)((Rwave2 >= prob ) && (*g=d));
	};
	*g = *g+1;
	prob = tex3D<myfloat>(*tex_prob, 0.5f, *nb+0.5f, t+0.5f);
	(void)((Rwave2 < prob ) && (*g = 0));
	(void)((Rwave2 == 1)    && (*g = nQ-1));
#endif

}
__device__ __forceinline__ void wave_find(int *nb, int *g, int *countnb, int *countg, int *idx_nb, Count *cnt)
{
	int tmp=1;
	while(tmp)
	{
		if(*countnb < cnt->nb_cnt[idx_nb[*nb]])
		{
			*countnb+=1;
			tmp=0;
		}
		else
		{
			*nb +=1;
			*countnb=0;
			*countg=0;
			*g=0;
			tmp=1;
		}
	};
	while(!tmp)
	{
		if(*countg < cnt->g_cnt[*g][idx_nb[*nb]])
		{
			*countg+=1;
			tmp=1;
		}
		else
		{
			*g +=1;
			*countg=0;
			tmp=0;
		}
	};
}


struct PAIR
{
	int pos;
	myfloat val;
};

bool compare(PAIR p1, PAIR p2) {return p1.val < p2.val;}

void sort_idx(NarrowBand *narrBand)
{
	vector<PAIR> p(nB);

	for (int nb = 0; nb<nB; nb++)
	{
		p[nb].pos = nb;
		p[nb].val = narrBand[nb].kavg;
	}

	sort( p.begin(), p.end(), compare );

	for (int nb = 0; nb<nB; nb++)
	{
		narrBand[nb].idx = p[nb].pos;
	}
}

__global__ void kernel_results(myfloat *var, myfloat *device, int n, int ns, int stream)
{

	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	volatile int tx = threadIdx.x;

	for (int idx = tid; idx < n; idx += blockDim.x * gridDim.x)
	{
		device[idx] = solution[stream][idx];
		var[idx]    = variance[stream][idx];
	}
}
